#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#define N 2048

__global__ void add(float *a, float *b, float *c) {
	c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

void random_floats(float* x, int size)
{
	int i;
  float max = 10000.0;
	for (i=0;i<size;i++) {
		x[i]=((float)rand()/(float)(RAND_MAX))*max;
	}
}

int main(void) 
{ clock_t start, end;
  start = clock();
	float *a, *b, *c;
	float *d_a, *d_b, *d_c; 
  int flt_num = 100000;
	int size =  flt_num * sizeof(float);

  hipEvent_t start1, stop1;
  hipEventCreate(&start1);
  hipEventCreate(&stop1);

  hipEventRecord(start1);
	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);
  hipEventRecord(stop1);
  hipEventSynchronize(stop1);
	a = (float *)malloc(size); 
  random_floats(a, flt_num);
	b = (float *)malloc(size); 
  random_floats(b, flt_num);
	c = (float *)malloc(size);

  hipEvent_t start2, stop2;
  hipEventCreate(&start2);
  hipEventCreate(&stop2);

  hipEventRecord(start2);
	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
  hipEventRecord(stop2);
  hipEventSynchronize(stop2);

  hipEvent_t start3, stop3;
  hipEventCreate(&start3);
  hipEventCreate(&stop3);

  hipEventRecord(start3);
	add<<<N,32>>>(d_a, d_b, d_c);

  hipDeviceSynchronize();
	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
  hipEventRecord(stop3);
  hipEventSynchronize(stop3);

	for (int i=0;i<flt_num;i++) {
		printf("a[%d]=%f , b[%d]=%f, c[%d]=%f\n",i,a[i],i,b[i],i,c[i]);
	}

	free(a); free(b); free(c);
	hipFree(d_a); hipFree(d_b); hipFree(d_c);
  float msec_malloc = 0.0f;
  float msec_memcpy = 0.0f;
  float msec_execution = 0.0f;
  hipEventElapsedTime(&msec_malloc, start1, stop1);
  hipEventElapsedTime(&msec_memcpy, start2, stop2);
  hipEventElapsedTime(&msec_execution, start3, stop3);
  printf("Device malloc time : %.3f msec\n" ,msec_malloc);
  printf("Device memcpy time : %.3f msec\n", msec_memcpy);
  printf("Device execution time : %.3f msec\n", msec_execution);
  end = clock();
  printf("Execution time: %.3f msec\n", (double)(end - start) / CLOCKS_PER_SEC*1000);
      

	return 0;
}
