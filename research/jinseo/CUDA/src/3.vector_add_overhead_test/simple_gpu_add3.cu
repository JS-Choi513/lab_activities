#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#define N 32

__global__ void add(float *a, float *b, float *c, int numElements) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < numElements){
    c[i] = a[i] + b[i];
  }
 // printf("in GPU Calculation Result is... %.3f\n",c[blockIdx.x]);
 // printf("a=%f , b=%f, c=%f\n",a[blockIdx.x],b[blockIdx.x],c[blockIdx.x]);

}

void random_floats(float* x, int size)
{
	int i;
  float max = 10000.0;
	for (i=0;i<size;i++) {
		x[i]=((float)rand()/(float)(RAND_MAX))*max;
	}
}

int main(void) 
{ clock_t start, end;
  start = clock();
	float *a, *b, *c;
	float *d_a, *d_b, *d_c; 
  int flt_num = 16384;
	int size =  flt_num * sizeof(float);

  hipEvent_t start1, stop1;
  hipEventCreate(&start1);
  hipEventCreate(&stop1);

  hipEventRecord(start1);
	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);
  hipEventRecord(stop1);
  hipEventSynchronize(stop1);
	a = (float *)malloc(size); 
  random_floats(a, flt_num);
	b = (float *)malloc(size); 
  random_floats(b, flt_num);
	c = (float *)malloc(size);

  hipEvent_t start2, stop2;
  hipEventCreate(&start2);
  hipEventCreate(&stop2);

  hipEventRecord(start2);
	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
  hipEventRecord(stop2);
  hipEventSynchronize(stop2);

  hipEvent_t start3, stop3;
  hipEventCreate(&start3);
  hipEventCreate(&stop3);

  hipEventRecord(start3);
	add<<<1,16384>>>(d_a, d_b, d_c, 16384);

  hipDeviceSynchronize();
	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
  hipEventRecord(stop3);
  hipEventSynchronize(stop3);

	for (int i=0;i<flt_num;i++) {
		printf("a[%d]=%f , b[%d]=%f, c[%d]=%f\n",i,a[i],i,b[i],i,c[i]);
	}

	free(a); free(b); free(c);
	hipFree(d_a); hipFree(d_b); hipFree(d_c);
  float msec_malloc = 0.0f;
  float msec_memcpy = 0.0f;
  float msec_execution = 0.0f;
  hipEventElapsedTime(&msec_malloc, start1, stop1);
  hipEventElapsedTime(&msec_memcpy, start2, stop2);
  hipEventElapsedTime(&msec_execution, start3, stop3);
  printf("Device malloc time : %.3f msec\n" ,msec_malloc);
  printf("Device memcpy time : %.3f msec\n", msec_memcpy);
  printf("Device execution time : %.3f msec\n", msec_execution);
  end = clock();
  printf("Execution time: %.3f msec\n", (double)(end - start) / CLOCKS_PER_SEC*1000);
      

	return 0;
}
